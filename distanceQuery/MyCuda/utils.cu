#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <hipcub/hipcub.hpp>

size_t utils::cubTempStorageBytes = 0;
void* utils::cubTempStorage = nullptr;

static void* cubTempStorage;
static size_t cubTempStorageBytes;


void utils::allocCubTemp()
{
	printf("CUB use mem: %d bytes\n", cubTempStorageBytes);
	checkCudaErrors(hipMalloc((void**)&cubTempStorage, cubTempStorageBytes * sizeof(int)));
}

template<typename T>
void utils::sortRegist(int* label, int* label_out, int num_items)
{
	T* d_in;
	T* d_out;
	void* storage = nullptr;
	size_t storage_bytes = 0;
	hipcub::DeviceRadixSort::SortPairs(cubTempStorage, cubTempStorageBytes, d_in, d_out, label, label_out, num_items);
	if (storage_bytes > cubTempStorageBytes) cubTempStorageBytes = storage_bytes;
}

template<typename T>
void utils::sort(T* d_in, T* d_out, int* label, int* label_out, int num_items)
{
	hipEvent_t start, stop;
	float elapsedTime = 0.0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipcub::DeviceRadixSort::SortPairs(cubTempStorage, cubTempStorageBytes, d_in, d_out, label, label_out, num_items);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%s time: %f ms\n", "sort", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

template<typename T>
void utils::minReduceRegist(int num_items) {
	T* d_in = nullptr;
	T* d_out = nullptr;
	void* storage = nullptr;
	size_t storage_bytes = 0;
	hipcub::DeviceReduce::Min(storage, storage_bytes, d_in, d_out, num_items);
	if (storage_bytes > cubTempStorageBytes) cubTempStorageBytes = storage_bytes;
}

template<typename T>
void utils::maxReduceRegist(int num_items) {
	T* d_in = nullptr;
	T* d_out = nullptr;
	void* storage = nullptr;
	size_t storage_bytes = 0;
	hipcub::DeviceReduce::Max(storage, storage_bytes, d_in, d_out, num_items);
	if (storage_bytes > cubTempStorageBytes) cubTempStorageBytes = storage_bytes;
}

template<typename T>
void utils::minReduce(T* d_in, T* d_out, int num_items)
{
	hipcub::DeviceReduce::Min(cubTempStorage, cubTempStorageBytes, d_in, d_out, num_items);
}

template<typename T>
void utils::maxReduce(T* d_in, T* d_out, int num_items)
{
	hipcub::DeviceReduce::Max(cubTempStorage, cubTempStorageBytes, d_in, d_out, num_items);
}

template<typename T>
float utils::minReduce(const std::string& name, T* d_in, T* d_out, int num_items)
{
	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	printf("reduce %d\n", num_items);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipcub::DeviceReduce::Min(cubTempStorage, cubTempStorageBytes, d_in, d_out, num_items);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%s time: %f ms\n", "min reduce", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsedTime;
}

template<typename T>
float utils::maxReduce(const std::string& name, T* d_in, T* d_out, int num_items)
{
	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	printf("reduce %d\n", num_items);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipcub::DeviceReduce::Max(cubTempStorage, cubTempStorageBytes, d_in, d_out, num_items);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%s time: %f ms\n", "min reduce", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsedTime;
}

template void utils::minReduce<float>(float*, float*, int);
template void utils::maxReduce<float>(float*, float*, int);
template float utils::minReduce<float>(const std::string&, float*, float*, int);
template float utils::maxReduce<float>(const std::string&, float*, float*, int);
template void utils::minReduceRegist<float>(int);
template void utils::maxReduceRegist<float>(int);